#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#define GL_GLEXT_PROTOTYPES
#include <hip/hip_runtime.h>
#include <GL/glut.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define rnd(x) (x*rand() / RAND_MAX)
#define SPEED 1.0f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define INF 2e10f

const int window_width = 1024;
const int window_height = 1024;

//The pixel buffer object id that is used to address the pixel buffer that
//is created on the GPU
GLuint gl_PBO;
//The cuda graphics resource that is used to link the OpenGL pixel buffer
//to CUDA
hipGraphicsResource *cuda_pbo_resource;
// the pointer used to store the GPU address to the pixel buffer object
// to give to the kernel for computation
texture<float,2> texConstSrc;
texture<float,2> texIn;
texture<float,2> texOut;

float *dev_inSrc;
float *dev_outSrc;
float *dev_constSrc;
uchar4 *d_dst = NULL;

dim3 blockSize;
dim3 gridSize;

__global__ void copy_const_kernel( float *iptr){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float c = tex2D(texConstSrc,x,y);
	if(c != 0){
		iptr[offset] = c;
	}
}

__global__ void transferHeat(float * out, bool dstOut){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float t, l, c, r, b, tl, tr, bl, br;
	if(dstOut){
		t = tex2D(texIn,x,y-1);
		l = tex2D(texIn,x-1, y);
		c = tex2D(texIn,x,y);
		r = tex2D(texIn,x+1, y);
		b = tex2D(texIn,x,y+1);
		tl = tex2D(texIn, x-1, y-1);
		tr = tex2D(texIn, x+1, y-1);
		bl = tex2D(texIn, x-1, y+1);
		br = tex2D(texIn, x+1, y+1);
	}else{
		t = tex2D(texOut,x,y-1);
		l = tex2D(texOut,x-1, y);
		c = tex2D(texOut,x,y);
		r = tex2D(texOut,x+1, y);
		b = tex2D(texOut,x,y+1);
		tl = tex2D(texOut, x-1, y-1);
		tr = tex2D(texOut, x+1, y-1);
		bl = tex2D(texOut, x-1, y+1);
		br = tex2D(texOut, x+1, y+1);
	}

	out[offset] = c + ((t + b + l + r + tl + tr + bl + br)/8 - c);


}

__global__ void float_to_color(uchar4 * pixels, float* in){
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;

	float num = in[offset];

	pixels[offset].x = (int)(num*255);
	pixels[offset].y = (int)(0);
	pixels[offset].z = (int)((MAX_TEMP-num) * 255);
	pixels[offset].w = 255;
}

void display(void){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	glFlush();

	hipGraphicsMapResources(1, &cuda_pbo_resource, NULL);

	hipEventRecord(start, 0);
	volatile bool dstOut = false;
	for(int i = 0; i < 200; i++){
		float *in, *out;
		if(dstOut){
			in = dev_inSrc;
			out = dev_outSrc;
		} else {
			out = dev_inSrc;
			in = dev_outSrc;
		}
		copy_const_kernel<<<gridSize, blockSize>>>(in);
		transferHeat<<<gridSize, blockSize>>>(out, dstOut);
		dstOut = !dstOut;
	}
	float_to_color<<<gridSize, blockSize>>>(d_dst, dev_inSrc);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);

	glDrawPixels(window_width, window_height, GL_RGBA, GL_UNSIGNED_BYTE, 0);

	glutSwapBuffers();

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	char title[20];
	sprintf(title,"time: %3.2f", elapsedTime/200);

	glutSetWindowTitle(title);

	hipEventDestroy(start);
	hipEventDestroy(stop);

}

void init(){
	blockSize.x = 16;
	blockSize.y = 16;
	blockSize.z = 1;

	gridSize.x = window_width/16;
	gridSize.y = window_height/16;
	gridSize.z = 1;

	glGenBuffers(1, &gl_PBO);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, window_width*window_height*sizeof(uchar4), NULL, GL_DYNAMIC_DRAW_ARB);

	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, gl_PBO, cudaGraphicsMapFlagsNone);
	hipGraphicsMapResources(1, &cuda_pbo_resource, NULL);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&d_dst, &num_bytes, cuda_pbo_resource);

	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);

	hipMalloc((void**)&dev_inSrc, window_width*window_height*sizeof(float));
	hipMalloc((void**)&dev_outSrc, window_width*window_height*sizeof(float));
	hipMalloc((void**)&dev_constSrc, window_width*window_height*sizeof(float));

	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();

	hipBindTexture2D(NULL, texConstSrc, dev_constSrc, desc, window_width, window_height, sizeof(float)*window_width);
	hipBindTexture2D(NULL, texIn, dev_inSrc, desc, window_width, window_height, sizeof(float)*window_width);
	hipBindTexture2D(NULL, texOut, dev_outSrc, desc, window_width, window_height, sizeof(float)*window_width);

	float *temp = (float*)malloc(window_width*window_height*sizeof(float));
	for(int i = 0; i < window_width*window_height; i++){
		temp[i] = 0;
		int x = i % window_width;
		int y = i / window_height;
		if((x > 300) && (x < 600) && (y > 310) && (y < 601)){
			temp[i] = MAX_TEMP;
		}
	}
	temp[window_width*100+100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[window_width*700+100] = MIN_TEMP;
	temp[window_width*300+300] = MIN_TEMP;
	temp[window_width*200+700] = MIN_TEMP;
	for(int y=800; y < 900; y++){
		for(int x=400; x<500; x++){
			temp[x+y*window_width] = MIN_TEMP;
		}
	}
	hipMemcpy(dev_constSrc, temp, window_width*window_height*sizeof(float), hipMemcpyHostToDevice);
	for(int i = 0; i < window_width*window_height; i++){
		temp[i] = MIN_TEMP;
	}
	hipMemcpy(dev_inSrc, temp, window_width*window_height*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_outSrc, dev_inSrc, window_width*window_height*sizeof(float), hipMemcpyDeviceToDevice);
	free(temp);

}

void deInit(){
	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);
	glDeleteBuffers(1, &gl_PBO);

	hipUnbindTexture(texIn);
	hipUnbindTexture(texOut);
	hipUnbindTexture(texConstSrc);

	hipFree(dev_inSrc);
	hipFree(dev_outSrc);
	hipFree(dev_constSrc);
}

void keyboard(unsigned char key, int x, int y){
	switch(key){
	case ' ':;
		glutPostRedisplay();
		break;
	}
}

void idle(void){
	glutPostRedisplay();
}

int main(int argc, char** argv) {
	hipDeviceProp_t prop;
	int dev;

	memset(&prop, 0, sizeof(hipDeviceProp_t));
	prop.major = 1;
	prop.minor = 0;
	hipChooseDevice(&dev, &prop);
	cudaGLSetGLDevice(dev);

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
	glutInitWindowSize(window_width, window_height);
	glutCreateWindow("Ray Tracing Renderer");
	glutIdleFunc(idle);
	glutDisplayFunc(display);
	glutKeyboardFunc(keyboard);

	init();
	glutMainLoop();
	deInit();

	return 0;
}
