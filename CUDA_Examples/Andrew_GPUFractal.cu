#include "hip/hip_runtime.h"
//nvcc GPUFractal.cu -o GPUFrac -lGLEW -lglut -lm -lGLU -lGL
#define GL_GLEXT_PROTOTYPES

#include <GL/glew.h>
#include <GL/freeglut.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define MIN(a,b) ((a) < (b) ? (a) : (b))

#define BUFFER_DATA(i) ((char *)0 + i)

unsigned int window_width = 1024;
unsigned int window_height = 1024;

float dt;

int frameCount = 0;
int currentTime = 0;
int previousTime = 0;

float *pixels;
float *pixels_gpu;

float ab[2] = {0,0};
float offset[2] = {0,0};
float scale = 2;
unsigned int loops = 500;

GLuint gl_PBO, gl_Tex, gl_Shader;
struct hipGraphicsResource *cuda_pbo_resource;

uchar4 *h_Src = 0;
uchar4 *d_dst = NULL;

static const char *shader_code =
    "!!ARBfp1.0\n"
    "TEX result.color, fragment.texcoord, texture[0], 2D; \n"
    "END";

__device__ float ab_gpu[2];
__device__ float offset_gpu[2];
__device__ float scale_gpu;
__device__ unsigned int loops_gpu;

__device__ unsigned char fractal(){
	int i = 0;
	float a = ab_gpu[0];
	float b = ab_gpu[1];
	float scale = scale_gpu;
	float xOffset = offset_gpu[1];
	float yOffset = offset_gpu[0];
	float x,y;
	x = (((float)threadIdx.x / (float)blockDim.x) * (scale_gpu*2)) - scale + xOffset;
	y = (((float)blockIdx.x / (float)gridDim.x) * (scale_gpu*2)) - scale + yOffset;
	float dist = 10000000000000;
	while(i < loops_gpu){
		float tmpX = (x*x) - (y*y) + a;
		y = 2*x*y + b;
		x = tmpX;
		if(x*x + y*y > 4){
			i=0;
			break;
		}
		i++;
		dist = min(dist, x*x + y*y);
	}

	return (unsigned char)((float)i/((float)loops_gpu)*512*sqrt(dist));
}

__global__ void fractalSetPixels(uchar4 *pixels_gpu){
	int tid = (threadIdx.x) + blockIdx.x * (blockDim.x);

	int contrast = fractal();

	pixels_gpu[tid].x = 0;
	pixels_gpu[tid].y = contrast;
	pixels_gpu[tid].z = 0;
	pixels_gpu[tid].w = 0;
}

void computeFPS(){

	frameCount++;

	currentTime = glutGet(GLUT_ELAPSED_TIME);

	int timeInterval = currentTime - previousTime;

	if(timeInterval > 1000){
		char fps[256];

		sprintf(fps, "%3.1f fps", frameCount / (timeInterval / 1000.f));

		glutSetWindowTitle(fps);

		previousTime = currentTime;

		frameCount = 0;
	}
}

int oldTimeSinceStart = 0;
void display(void)
{
	int timeSinceStart = glutGet(GLUT_ELAPSED_TIME);
	dt = (float)(timeSinceStart - oldTimeSinceStart)/1000.0f;
	oldTimeSinceStart = timeSinceStart;
	glFlush();

	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&d_dst, &num_bytes, cuda_pbo_resource);

	fractalSetPixels<<<window_width, window_height>>>(d_dst);

	hipDeviceSynchronize();

	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);

	glBindTexture(GL_TEXTURE_2D, gl_Tex);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, window_width, window_height, GL_RGBA, GL_UNSIGNED_BYTE, BUFFER_DATA(0));

	glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, gl_Shader);
	glEnable(GL_FRAGMENT_PROGRAM_ARB);
	glDisable(GL_DEPTH_TEST);

	glBegin(GL_QUADS);
	glTexCoord2f(0.0f, 0.0f);
	glVertex2f(-1.0f, -1.0f);
	glTexCoord2f(1.0f, 0.0f);
	glVertex2f(1.0f, -1.0f);
	glTexCoord2f(1.0f, 1.0f);
	glVertex2f(1.0f, 1.0f);
	glTexCoord2f(0.0f, 1.0f);
	glVertex2f(-1.0f, 1.0f);
	glEnd();

	glBindTexture(GL_TEXTURE_2D, 0);
	glDisable(GL_FRAGMENT_PROGRAM_ARB);


	//hipMemcpy(pixels, pixels_gpu, window_width*window_height*3*sizeof(float), hipMemcpyDeviceToHost);
	//glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels_gpu);
	glutSwapBuffers();

	computeFPS();
}

void reset(){
	scale = 2;
	offset[0] = 0;
	offset[1] = 0;
	ab[0] = -0.747753;
	ab[1] = 0.05f;

	hipMemcpyToSymbol(HIP_SYMBOL(scale_gpu), &scale, sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(offset_gpu), offset, 2*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(ab_gpu), ab, 2*sizeof(float), 0, hipMemcpyHostToDevice);
}

void keyboard(unsigned char key, int x, int y){
	switch(key){
	case 'w':
		ab[0] += 0.1f * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(ab_gpu), ab, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case 's':
		ab[0] -= 0.1f * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(ab_gpu), ab, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case 'a':
		ab[1] -= 0.1f * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(ab_gpu), ab, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case 'd':
		ab[1] += 0.1f * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(ab_gpu), ab, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case 'q':
		scale -= 0.5f * scale * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(scale_gpu), &scale, sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case 'e':
		scale += 0.5f * scale * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(scale_gpu), &scale, sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case 'r':
		reset();
		break;
	case ' ':
		printf("===============================");
		printf("\na = %f, b = %f\n", ab[0], ab[1]);
		printf("scale = %f\n", scale);
		printf("offset X = %f \t offset Y = %f\n", offset[1], offset[0]);
		printf("max iterations = %d\n", loops);
		printf("===============================\n");
		break;
	}

	//glutPostRedisplay();
}

void specialInput(int key, int x, int Y){
	switch(key){
	case GLUT_KEY_UP:
		offset[0] += 0.1f * scale * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(offset_gpu), offset, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case GLUT_KEY_DOWN:
		offset[0] -= 0.1f * scale * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(offset_gpu), offset, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case GLUT_KEY_LEFT:
		offset[1] -= 0.1f * scale * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(offset_gpu), offset, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case GLUT_KEY_RIGHT:
		offset[1] += 0.1f * scale * dt;
		hipMemcpyToSymbol(HIP_SYMBOL(offset_gpu), offset, 2*sizeof(float), 0, hipMemcpyHostToDevice);
		break;
	case GLUT_KEY_PAGE_UP:
		loops += 100;
		hipMemcpyToSymbol(HIP_SYMBOL(loops_gpu), &loops, sizeof(int), 0, hipMemcpyHostToDevice);
		break;
	case GLUT_KEY_PAGE_DOWN:
		if(loops - 100 > 0){
			loops -= 100;
		}
		hipMemcpyToSymbol(HIP_SYMBOL(loops_gpu), &loops, sizeof(int), 0, hipMemcpyHostToDevice);
		break;
	}
	//glutPostRedisplay();
}

GLuint compileASMShader(GLenum program_type, const char *code){
	GLuint program_id;
	glGenProgramsARB(1, &program_id);
	glBindProgramARB(program_type, program_id);
	glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB, (GLsizei) strlen(code), (GLubyte *) code);

	GLint error_pos;
	glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);

	if(error_pos != -1){
		const GLubyte *error_string;
		error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
		fprintf(stderr, "Program error at position: %d\b%s\n", (int)error_pos, error_string);
		return 0;
	}

	return program_id;
}

void init(){

	if (h_Src)
	{
		free(h_Src);
		h_Src = 0;
	}

	if (gl_Tex)
	{
		glDeleteTextures(1, &gl_Tex);
		gl_Tex = 0;
	}

	if (gl_PBO)
	{
		hipGraphicsUnregisterResource(cuda_pbo_resource);
		glDeleteBuffers(1, &gl_PBO);
		gl_PBO = 0;
	}

	h_Src = (uchar4 *)malloc(window_width * window_height * 4);

	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &gl_Tex);
	glBindTexture(GL_TEXTURE_2D, gl_Tex);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, window_width, window_height, 0, GL_RGBA, GL_UNSIGNED_BYTE, h_Src);

	glGenBuffers(1, &gl_PBO);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
	glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, window_width * window_height * sizeof(uchar4), NULL, GL_STREAM_COPY);

	hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, gl_PBO, cudaGraphicsMapFlagsWriteDiscard);

	gl_Shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);


	hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void**)&d_dst, &num_bytes, cuda_pbo_resource);

	pixels = (float *)malloc(window_width*window_height*3*sizeof(float));
	hipMalloc((void**)&pixels_gpu, window_width*window_height*3*sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(ab_gpu), ab, 2*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(scale_gpu), &scale, sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(offset_gpu), offset, 2*sizeof(float), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(loops_gpu), &loops, sizeof(int), 0, hipMemcpyHostToDevice);
}

void deInit(){

	if(h_Src){
		free(h_Src);
		h_Src = 0;
	}

	free(pixels);
	hipFree(pixels_gpu);

	hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

	glDeleteBuffers(1, &gl_PBO);
	glDeleteTextures(1, &gl_Tex);
	glDeleteProgramsARB(1, &gl_Shader);
}

void idle(void){

	computeFPS();

	glutPostRedisplay();
}

int main(int argc, char** argv)
{
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
   	glutInitWindowSize(window_width, window_height);
   	glutCreateWindow("BitMap");
   	glutDisplayFunc(display);
   	glutKeyboardFunc(keyboard);
   	glutSpecialFunc(specialInput);
   	glutIdleFunc(idle);

   	printf("Loading extensions: %s\n", glewGetErrorString(glewInit()));

   	if (!glewIsSupported("GL_VERSION_1_5 GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object"))
   	    {
   	        fprintf(stderr, "Error: failed to get minimal extensions for demo\n");
   	        fprintf(stderr, "This sample requires:\n");
   	        fprintf(stderr, "  OpenGL version 1.5\n");
   	        fprintf(stderr, "  GL_ARB_vertex_buffer_object\n");
   	        fprintf(stderr, "  GL_ARB_pixel_buffer_object\n");
   	        exit(EXIT_SUCCESS);
   	    }
	init();
   	glutMainLoop();
   	deInit();
}
